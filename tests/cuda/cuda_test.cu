#include "hip/hip_runtime.h"

#define N 5

__global__ void add(int *a, int *b, int *c)
{
    int tID = blockIdx.x;
    if (tID < N)
    {
        c[tID] = a[tID] + b[tID];
    }
}

double foo()
{
    //   A function annotation. Opens region "function=foo" in Caliper,
    // and automatically closes it at the end of the function.

    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    hipMalloc((void **) &dev_a, N*sizeof(int));
    hipMalloc((void **) &dev_b, N*sizeof(int));
    hipMalloc((void **) &dev_c, N*sizeof(int));

    // Fill Arrays
    for (int i = 0; i < N; i++)
    {
        a[i] = i,
        b[i] = i;
    }
    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

    add<<<N,1>>>(dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

    return 0.5;
}

int main() {

    int iterations = 10;
    for (int i = 0; i < iterations; i++) {
        int out = foo();
    }  
}
